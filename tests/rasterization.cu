#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <iostream>

#include "helpers.cuh"
#include "tinyrend/rasterization/kernel.cuh"
#include "tinyrend/rasterization/primitives/image_gaussian.h"
// #include "tinyrend/rasterization/primitives/image_triangle.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

using namespace tinyrend::rasterization;

void save_png(float *buffer, int width, int height, const char *filename) {
    // Convert float buffer to unsigned char buffer
    unsigned char *image_data = new unsigned char[width * height];

    // Normalize and convert float values to 0-255 range
    for (int i = 0; i < width * height; i++) {
        // Clamp values between 0 and 1
        float value = std::max(0.0f, std::min(1.0f, buffer[i]));
        // Convert to 0-255 range
        image_data[i] = static_cast<unsigned char>(value * 255.0f);
    }

    // Save as PNG
    stbi_write_png(filename, width, height, 1, image_data, width);

    // Clean up
    delete[] image_data;
}

auto test_rasterization() -> int {
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    const int n_primitives = 2;

    // Create Some Image Gaussians on GPU
    glm::fvec2 *h_mu = new glm::fvec2[n_primitives];
    for (int i = 0; i < n_primitives; i++) {
        h_mu[i] = glm::fvec2(i, i) * 4.0f + 6.0f;
    }
    glm::fvec3 *h_conics = new glm::fvec3[n_primitives];
    for (int i = 0; i < n_primitives; i++) {
        h_conics[i] = glm::fvec3(0.25f, 0.0f, 0.25f);
    }
    float *h_features = new float[n_primitives * 3];
    for (int i = 0; i < n_primitives; i++) {
        h_features[i * 3 + 0] = i;
        h_features[i * 3 + 1] = i;
        h_features[i * 3 + 2] = i;
    }

    // Create Some Image Gaussians on GPU
    glm::fvec2 *d_mu = create_device_ptr(h_mu[0], n_primitives);
    glm::fvec3 *d_conics = create_device_ptr(h_conics[0], n_primitives);
    float *d_features = create_device_ptr(h_features[0], n_primitives * 3);

    ImageGaussians<3> primitives{};
    primitives.mu = d_mu;
    primitives.conics = d_conics;
    primitives.features = d_features;

    // Create isect info on GPU
    uint32_t isect_primitive_ids_host[n_primitives] = {0, 1};
    uint32_t isect_prefix_sum_per_tile_host[1] = {2};
    uint32_t *isect_primitive_ids;
    hipMalloc(&isect_primitive_ids, sizeof(uint32_t) * n_primitives);
    hipMemcpy(
        isect_primitive_ids,
        isect_primitive_ids_host,
        sizeof(uint32_t) * n_primitives,
        hipMemcpyHostToDevice
    );
    uint32_t *isect_prefix_sum_per_tile;
    hipMalloc(&isect_prefix_sum_per_tile, sizeof(uint32_t) * 1);
    hipMemcpy(
        isect_prefix_sum_per_tile,
        isect_prefix_sum_per_tile_host,
        sizeof(uint32_t) * 1,
        hipMemcpyHostToDevice
    );

    // image size
    const uint32_t image_h = 16;
    const uint32_t image_w = 16;

    // Create buffer for alpha values
    float *buffer_alpha;
    hipMalloc(&buffer_alpha, sizeof(float) * image_h * image_w);

    // Create buffer for features
    hipMalloc(&primitives.buffer_features, sizeof(float) * image_h * image_w * 3);

    // launch rasterization kernel
    dim3 threads(16, 16, 1);
    dim3 grid(1, 1, 1);
    size_t shmem_size = ImageGaussians<3>::shmem_size_per_primitive() * 16 * 16;
    rasterization<<<grid, threads, shmem_size>>>(
        primitives,
        image_h,
        image_w,
        isect_primitive_ids,
        isect_prefix_sum_per_tile,
        buffer_alpha,
        nullptr // buffer_last_primitive_id
    );

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    // copy buffer_alpha back to host
    float *buffer_alpha_host = new float[image_h * image_w];
    hipMemcpy(
        buffer_alpha_host,
        buffer_alpha,
        sizeof(float) * image_h * image_w,
        hipMemcpyDeviceToHost
    );

    // print buffer_alpha
    for (int i = 0; i < image_h; i++) {
        for (int j = 0; j < image_w; j++) {
            printf("%f ", buffer_alpha_host[i * image_w + j]);
        }
        printf("\n");
    }

    // save buffer_alpha_host into a png file
    save_png(buffer_alpha_host, image_w, image_h, "buffer_alpha.png");

    return 0;
}

// auto test_rasterization2() -> int {
//     hipError_t err = hipSetDevice(0);
//     if (err != hipSuccess) {
//         printf("CUDA Error: %s\n", hipGetErrorString(err));
//     }

//     const int n_primitives = 2;

//     // Create Some Image Triangles on GPU
//     glm::fvec2 *h_v0 = new glm::fvec2[n_primitives];
//     for (int i = 0; i < n_primitives; i++) {
//         h_v0[i] = glm::fvec2(i, i) * 4.0f + 0.0f;
//     }
//     glm::fvec2 *h_v1 = new glm::fvec2[n_primitives];
//     for (int i = 0; i < n_primitives; i++) {
//         h_v1[i] = glm::fvec2(i + 4, i) * 4.0f + 0.0f;
//     }
//     glm::fvec2 *h_v2 = new glm::fvec2[n_primitives];
//     for (int i = 0; i < n_primitives; i++) {
//         h_v2[i] = glm::fvec2(i, i + 4) * 4.0f + 0.0f;
//     }

//     // Create Some Image Triangles on GPU
//     glm::fvec2 *d_v0 = create_device_ptr(h_v0[0], n_primitives);
//     glm::fvec2 *d_v1 = create_device_ptr(h_v1[0], n_primitives);
//     glm::fvec2 *d_v2 = create_device_ptr(h_v2[0], n_primitives);

//     ImageTriangles primitives{};
//     primitives.v0 = d_v0;
//     primitives.v1 = d_v1;
//     primitives.v2 = d_v2;

//     // Create isect info on GPU
//     uint32_t isect_primitive_ids_host[n_primitives] = {0, 1};
//     uint32_t isect_prefix_sum_per_tile_host[1] = {2};
//     uint32_t *isect_primitive_ids;
//     hipMalloc(&isect_primitive_ids, sizeof(uint32_t) * n_primitives);
//     hipMemcpy(
//         isect_primitive_ids,
//         isect_primitive_ids_host,
//         sizeof(uint32_t) * n_primitives,
//         hipMemcpyHostToDevice
//     );
//     uint32_t *isect_prefix_sum_per_tile;
//     hipMalloc(&isect_prefix_sum_per_tile, sizeof(uint32_t) * 1);
//     hipMemcpy(
//         isect_prefix_sum_per_tile,
//         isect_prefix_sum_per_tile_host,
//         sizeof(uint32_t) * 1,
//         hipMemcpyHostToDevice
//     );

//     // image size
//     const uint32_t image_h = 16;
//     const uint32_t image_w = 16;

//     // Create buffer for alpha values
//     float *buffer_alpha;
//     hipMalloc(&buffer_alpha, sizeof(float) * image_h * image_w);

//     // launch rasterization kernel
//     dim3 threads(16, 16, 1);
//     dim3 grid(1, 1, 1);
//     size_t shmem_size = ImageTriangles::shmem_size_per_primitive() * 16 * 16;
//     rasterization<<<grid, threads, shmem_size>>>(
//         primitives,
//         image_h,
//         image_w,
//         isect_primitive_ids,
//         isect_prefix_sum_per_tile,
//         buffer_alpha,
//         nullptr // buffer_last_primitive_id
//     );

//     err = hipGetLastError();
//     if (err != hipSuccess) {
//         printf("CUDA Error: %s\n", hipGetErrorString(err));
//     }
//     err = hipDeviceSynchronize();
//     if (err != hipSuccess) {
//         printf("CUDA Error: %s\n", hipGetErrorString(err));
//     }

//     // copy buffer_alpha back to host
//     float *buffer_alpha_host = new float[image_h * image_w];
//     hipMemcpy(
//         buffer_alpha_host,
//         buffer_alpha,
//         sizeof(float) * image_h * image_w,
//         hipMemcpyDeviceToHost
//     );

//     // print buffer_alpha
//     for (int i = 0; i < image_h; i++) {
//         for (int j = 0; j < image_w; j++) {
//             printf("%f ", buffer_alpha_host[i * image_w + j]);
//         }
//         printf("\n");
//     }

//     // save buffer_alpha_host into a png file
//     save_png(buffer_alpha_host, image_w, image_h, "buffer_alpha.png");

//     return 0;
// }

auto main() -> int {
    int fails = 0;
    fails += test_rasterization();

    if (fails == 0) {
        printf("\nAll tests passed!\n");
    } else {
        printf("\n%d tests failed!\n", fails);
    }

    return fails;
}