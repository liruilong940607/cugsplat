#include <hip/hip_runtime.h>
#include <iostream>

#include "helpers.cuh"
#include "helpers.h"
#include "tinyrend/core/vec.h"
#include "tinyrend/rasterization/kernel.cuh"
#include "tinyrend/rasterization/operators/image_gaussian.cuh"
#include "tinyrend/rasterization/operators/simple_planer.cuh"

using namespace tinyrend;
using namespace tinyrend::rasterization;

auto test_rasterization_simple_planer() -> int {

    // Configurations
    const int n_primitives = 2;
    const uint32_t image_height = 28;
    const uint32_t image_width = 22;
    const uint32_t tile_width = 8;
    const uint32_t tile_height = 16;
    dim3 threads(tile_width, tile_height, 1);
    dim3 grid(1, 1, 1);

    // Create primitive data:
    auto const opacity_ptr = create_device_ptr<float>({0.5f, 0.7f});
    // Create isect info: all two primitives are intersected with the first tile
    auto const isect_primitive_ids = create_device_ptr<uint32_t>({0, 1});
    auto const isect_prefix_sum_per_tile = create_device_ptr<uint32_t>({2});

    // Prepare forward outputs
    auto render_alpha_ptr =
        create_device_ptr<float>(image_height * image_width); // only alloc mem, no init

    // Create forward operator
    SimplePlanerRasterizeKernelForwardOperator forward_op{};
    forward_op.opacity_ptr = opacity_ptr;
    forward_op.render_alpha_ptr = render_alpha_ptr;

    // Launch forward rasterization
    size_t forward_sm_size =
        decltype(forward_op)::sm_size_per_primitive() * threads.x * threads.y;
    rasterize_kernel<<<grid, threads, forward_sm_size>>>(
        forward_op,
        image_height,
        image_width,
        isect_primitive_ids,
        isect_prefix_sum_per_tile
    );

    // Copy data back to host and check the result
    auto const h_render_alpha_ptr =
        device_ptr_to_host_ptr<float>(render_alpha_ptr, image_height * image_width);
    for (int x = 0; x < tile_width; x++) {
        for (int y = 0; y < tile_height; y++) {
            int i = x + y * image_width;
            assert(is_close(h_render_alpha_ptr[i], 0.5f + (1 - 0.5f) * 0.7f));
        }
    }
    save_png(h_render_alpha_ptr, image_width, image_height, "results/render_alpha.png");

    // Prepare backward gradients
    auto const v_render_alpha_ptr =
        create_device_ptr<float>(image_height * image_width, 0.3f);
    auto v_opacity_ptr = create_device_ptr<float>(n_primitives, 0.0f); // zero init

    // Create backward operator
    SimplePlanerRasterizeKernelBackwardOperator backward_op{};
    backward_op.opacity_ptr = opacity_ptr;
    backward_op.render_alpha_ptr = render_alpha_ptr;
    backward_op.v_render_alpha_ptr = v_render_alpha_ptr;
    backward_op.v_opacity_ptr = v_opacity_ptr;

    // Launch backward rasterization
    size_t backward_sm_size =
        decltype(backward_op)::sm_size_per_primitive() * threads.x * threads.y;
    rasterize_kernel<<<grid, threads, backward_sm_size>>>(
        backward_op,
        image_height,
        image_width,
        isect_primitive_ids,
        isect_prefix_sum_per_tile,
        true // reverse order
    );

    // o = a + (1 - a) * b
    // o = 0.5f + (1 - 0.5f) * 0.7f
    // dl/da = dl/do * do/da = 0.3f * (1 - 0.7f) = 0.09f
    // dl/db = dl/do * do/db = 0.3f * 0.5f = 0.15f
    auto const h_v_opacity_ptr =
        device_ptr_to_host_ptr<float>(v_opacity_ptr, n_primitives);
    assert(is_close(h_v_opacity_ptr[0], 0.09f * tile_width * tile_height));
    assert(is_close(h_v_opacity_ptr[1], 0.15f * tile_width * tile_height));

    check_cuda_error();
    return 0;
}

auto test_rasterization_image_gaussian() -> int {

    // Configurations
    const int n_primitives = 2;
    const uint32_t image_height = 28;
    const uint32_t image_width = 22;
    const uint32_t tile_width = 8;
    const uint32_t tile_height = 16;
    dim3 threads(tile_width, tile_height, 1);
    dim3 grid(1, 1, 1);
    const int feature_dim = 1;

    // Create primitive data:
    auto const opacity_ptr = create_device_ptr<float>({0.5f, 0.7f});
    auto const mean_ptr =
        create_device_ptr<fvec2>({fvec2(3.0f, 3.0f), fvec2(4.0f, 4.0f)});
    auto const conic_ptr =
        create_device_ptr<fvec3>({fvec3(0.25f, 0.0f, 0.25f), fvec3(0.25f, 0.0f, 0.25f)}
        );
    auto const feature_ptr = create_device_ptr<float>({0.2f, 0.5f});
    // Create isect info: all two primitives are intersected with the first tile
    auto const isect_primitive_ids = create_device_ptr<uint32_t>({0, 1});
    auto const isect_prefix_sum_per_tile = create_device_ptr<uint32_t>({2});

    // Prepare forward outputs
    auto render_last_index_ptr = create_device_ptr<int32_t>(image_height * image_width);
    auto render_alpha_ptr =
        create_device_ptr<float>(image_height * image_width); // only alloc mem, no init
    auto render_feature_ptr = create_device_ptr<float>(
        image_height * image_width * feature_dim
    ); // only alloc mem, no init

    // Create forward operator
    ImageGaussianRasterizeKernelForwardOperator<feature_dim> forward_op{};
    forward_op.opacity_ptr = opacity_ptr;
    forward_op.mean_ptr = mean_ptr;
    forward_op.conic_ptr = conic_ptr;
    forward_op.feature_ptr = feature_ptr;
    forward_op.render_last_index_ptr = render_last_index_ptr;
    forward_op.render_alpha_ptr = render_alpha_ptr;
    forward_op.render_feature_ptr = render_feature_ptr;

    // Launch forward rasterization
    size_t forward_sm_size =
        decltype(forward_op)::sm_size_per_primitive() * threads.x * threads.y;
    rasterize_kernel<<<grid, threads, forward_sm_size>>>(
        forward_op,
        image_height,
        image_width,
        isect_primitive_ids,
        isect_prefix_sum_per_tile
    );

    // Copy data back to host and check the result
    auto const h_render_alpha_ptr =
        device_ptr_to_host_ptr<float>(render_alpha_ptr, image_height * image_width);
    auto const h_render_last_index_ptr = device_ptr_to_host_ptr<int32_t>(
        render_last_index_ptr, image_height * image_width
    );
    auto const h_render_feature_ptr = device_ptr_to_host_ptr<float>(
        render_feature_ptr, image_height * image_width * feature_dim
    );
    save_png(h_render_alpha_ptr, image_width, image_height, "results/render_alpha.png");

    // Prepare backward gradients
    auto const v_render_alpha_ptr =
        create_device_ptr<float>(image_height * image_width, 0.3f);
    auto v_render_feature_ptr =
        create_device_ptr<float>(image_height * image_width * feature_dim, 0.2f);
    auto v_opacity_ptr = create_device_ptr<float>(n_primitives, 0.0f);  // zero init
    auto v_mean_ptr = create_device_ptr<fvec2>(n_primitives, fvec2{});  // zero init
    auto v_conic_ptr = create_device_ptr<fvec3>(n_primitives, fvec3{}); // zero init
    auto v_feature_ptr = create_device_ptr<float>(n_primitives, 0.0f);  // zero init

    // Create backward operator
    ImageGaussianRasterizeKernelBackwardOperator<feature_dim> backward_op{};
    backward_op.opacity_ptr = opacity_ptr;
    backward_op.mean_ptr = mean_ptr;
    backward_op.conic_ptr = conic_ptr;
    backward_op.feature_ptr = feature_ptr;
    backward_op.render_last_index_ptr = render_last_index_ptr;
    backward_op.render_alpha_ptr = render_alpha_ptr;
    backward_op.v_render_alpha_ptr = v_render_alpha_ptr;
    backward_op.v_render_feature_ptr = v_render_feature_ptr;
    backward_op.v_opacity_ptr = v_opacity_ptr;
    backward_op.v_mean_ptr = v_mean_ptr;
    backward_op.v_conic_ptr = v_conic_ptr;
    backward_op.v_feature_ptr = v_feature_ptr;

    // // Launch backward rasterization
    // size_t backward_sm_size =
    //     decltype(backward_op)::sm_size_per_primitive() * threads.x * threads.y;
    // rasterize_kernel<<<grid, threads, backward_sm_size>>>(
    //     backward_op,
    //     image_height,
    //     image_width,
    //     isect_primitive_ids,
    //     isect_prefix_sum_per_tile,
    //     true // reverse order
    // );

    // // o = a + (1 - a) * b
    // // o = 0.5f + (1 - 0.5f) * 0.7f
    // // dl/da = dl/do * do/da = 0.3f * (1 - 0.7f) = 0.09f
    // // dl/db = dl/do * do/db = 0.3f * 0.5f = 0.15f
    // auto const h_v_opacity_ptr =
    //     device_ptr_to_host_ptr<float>(v_opacity_ptr, n_primitives);
    // assert(is_close(h_v_opacity_ptr[0], 0.09f * tile_width * tile_height));
    // assert(is_close(h_v_opacity_ptr[1], 0.15f * tile_width * tile_height));

    check_cuda_error();
    return 0;
}

auto main() -> int {
    int fails = 0;
    fails += test_rasterization_simple_planer();
    fails += test_rasterization_image_gaussian();

    if (fails == 0) {
        printf("\nAll tests passed!\n");
    } else {
        printf("\n%d tests failed!\n", fails);
    }

    return fails;
}