#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <glm/glm.hpp>
#include <iostream>

#include "helpers.cuh"
#include <tinyrend/rasterization.cuh>

using namespace tinyrend;

struct ImageGaussians : public BasePrimitives<ImageGaussians> {
    /*
    A collection of 2D Gaussian primitives.
    */

    // Pointers to the device memory
    glm::fvec2 *mu;     // [N, 2]
    glm::fvec3 *conics; // [N, 3]

    __device__ bool initialize(
        uint32_t image_id,
        uint32_t pixel_x,
        uint32_t pixel_y,
        void *shmem_ptr,
        uint32_t shmem_n_primitives
    ) {
        _image_id = image_id;
        _pixel_x = pixel_x;
        _pixel_y = pixel_y;
        _shmem_ptr = shmem_ptr;
        _shmem_n_primitives = shmem_n_primitives;
        return true;
    }

    __device__ void load_to_shared_memory(uint32_t shmem_id, uint32_t global_id) {
        glm::fvec2 *shmem_ptr_mu = reinterpret_cast<glm::fvec2 *>(_shmem_ptr);
        glm::fvec3 *shmem_ptr_conics =
            reinterpret_cast<glm::fvec3 *>(&shmem_ptr_mu[_shmem_n_primitives]);
        shmem_ptr_mu[shmem_id] = mu[global_id];
        shmem_ptr_conics[shmem_id] = conics[global_id];
    }

    __device__ float get_light_attenuation(uint32_t shmem_id) {
        glm::fvec2 *shmem_ptr_mu = reinterpret_cast<glm::fvec2 *>(_shmem_ptr);
        glm::fvec3 *shmem_ptr_conics =
            reinterpret_cast<glm::fvec3 *>(&shmem_ptr_mu[_shmem_n_primitives]);
        auto const mu = shmem_ptr_mu[shmem_id];
        auto const conic = shmem_ptr_conics[shmem_id];

        auto const dx = _pixel_x - mu.x;
        auto const dy = _pixel_y - mu.y;
        auto const sigma =
            0.5f * (conic.x * dx * dx + conic.z * dy * dy) + conic.y * dx * dy;
        return exp(-sigma);
    }

    // private:
    uint32_t _image_id;
    uint32_t _pixel_x;
    uint32_t _pixel_y;
    void *_shmem_ptr;
    uint32_t _shmem_n_primitives;
};

auto test_rasterization() -> int {
    hipError_t err = hipSetDevice(0);
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    const int n_primitives = 2;

    // Create Some Image Gaussians on GPU
    glm::fvec2 *h_mu = new glm::fvec2[n_primitives];
    for (int i = 0; i < n_primitives; i++) {
        h_mu[i] = glm::fvec2(i, i);
    }
    glm::fvec3 *h_conics = new glm::fvec3[n_primitives];
    for (int i = 0; i < n_primitives; i++) {
        h_conics[i] = glm::fvec3(1.0f, 0.0f, 1.0f);
    }

    // Create Some Image Gaussians on GPU
    glm::fvec2 *d_mu = create_device_ptr(h_mu[0], n_primitives);
    glm::fvec3 *d_conics = create_device_ptr(h_conics[0], n_primitives);

    ImageGaussians primitives{};
    primitives.mu = d_mu;
    primitives.conics = d_conics;

    // Create isect info on GPU
    uint32_t isect_primitive_ids_host[n_primitives] = {0, 1};
    uint32_t isect_prefix_sum_per_tile_host[1] = {2};
    uint32_t *isect_primitive_ids;
    hipMalloc(&isect_primitive_ids, sizeof(uint32_t) * n_primitives);
    hipMemcpy(
        isect_primitive_ids,
        isect_primitive_ids_host,
        sizeof(uint32_t) * n_primitives,
        hipMemcpyHostToDevice
    );
    uint32_t *isect_prefix_sum_per_tile;
    hipMalloc(&isect_prefix_sum_per_tile, sizeof(uint32_t) * 1);
    hipMemcpy(
        isect_prefix_sum_per_tile,
        isect_prefix_sum_per_tile_host,
        sizeof(uint32_t) * 1,
        hipMemcpyHostToDevice
    );

    // image size
    const uint32_t image_h = 4;
    const uint32_t image_w = 4;

    // Create buffer for alpha values
    float *buffer_alpha;
    hipMalloc(&buffer_alpha, sizeof(float) * image_h * image_w);

    // launch rasterization kernel
    dim3 threads(16, 16, 1);
    dim3 grid(1, 1, 1);
    size_t shmem_size = (sizeof(glm::fvec2) + sizeof(glm::fvec3)) * 16 * 16;
    rasterization<<<grid, threads, shmem_size>>>(
        primitives,
        image_h,
        image_w,
        isect_primitive_ids,
        isect_prefix_sum_per_tile,
        buffer_alpha
    );

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    // copy buffer_alpha back to host
    float *buffer_alpha_host = new float[image_h * image_w];
    hipMemcpy(
        buffer_alpha_host,
        buffer_alpha,
        sizeof(float) * image_h * image_w,
        hipMemcpyDeviceToHost
    );

    // print buffer_alpha
    for (int i = 0; i < image_h; i++) {
        for (int j = 0; j < image_w; j++) {
            printf("%f ", buffer_alpha_host[i * image_w + j]);
        }
        printf("\n");
    }

    return 0;
}

auto main() -> int {
    int fails = 0;
    fails += test_rasterization();

    if (fails == 0) {
        printf("\nAll tests passed!\n");
    } else {
        printf("\n%d tests failed!\n", fails);
    }

    return fails;
}