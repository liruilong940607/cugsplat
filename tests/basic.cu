#include <hip/hip_runtime.h>
#include <iostream>

#include "preprocess/camera/simple_pinhole_ewa.cuh"
#include "preprocess/primitive_in/world3dgs.cuh"
#include "preprocess/primitive_out/image2dgs.cuh"
#include "preprocess/kernel.cuh"

int main(){
    // create camera
    auto const focal_length = glm::fvec2(800.0f, 600.0f);
    auto const principal_point = glm::fvec2(400.0f, 300.0f);
    auto const world_to_camera_R = glm::fmat3(1.0f);
    auto const world_to_camera_t = glm::fvec3(0.0f, 0.0f, 0.0f);

    cugsplat::preprocess::DeviceSimplePinholeCameraEWA d_camera;
    d_camera.n = 1;
    hipMalloc(&d_camera.focal_lengths, 1 * sizeof(glm::fvec2));
    hipMemcpy(d_camera.focal_lengths, &focal_length, 1 * sizeof(glm::fvec2), hipMemcpyHostToDevice);
    hipMalloc(&d_camera.principal_points, 1 * sizeof(glm::fvec2));
    hipMemcpy(d_camera.principal_points, &principal_point, 1 * sizeof(glm::fvec2), hipMemcpyHostToDevice);
    hipMalloc(&d_camera.world_to_cameras_R, 1 * sizeof(glm::fmat3));
    hipMemcpy(d_camera.world_to_cameras_R, &world_to_camera_R, 1 * sizeof(glm::fmat3), hipMemcpyHostToDevice);
    hipMalloc(&d_camera.world_to_cameras_t, 1 * sizeof(glm::fvec3));
    hipMemcpy(d_camera.world_to_cameras_t, &world_to_camera_t, 1 * sizeof(glm::fvec3), hipMemcpyHostToDevice);

    // create primitive input
    auto const opacity = float(0.8f);
    auto const mean = glm::fvec3(0.0f, 0.0f, 1.0f);
    auto const quat = glm::fvec4(1.0f, 0.0f, 0.0f, 0.0f);
    auto const scale = glm::fvec3(1.0f, 1.0f, 1.0f);

    cugsplat::preprocess::DevicePrimitiveInWorld3DGS d_gaussian_in;
    d_gaussian_in.n = 1;
    hipMalloc(&d_gaussian_in.opacities, 1 * sizeof(float));
    hipMemcpy(d_gaussian_in.opacities, &opacity, 1 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_gaussian_in.means, 1 * sizeof(glm::fvec3));
    hipMemcpy(d_gaussian_in.means, &mean, 1 * sizeof(glm::fvec3), hipMemcpyHostToDevice);
    hipMalloc(&d_gaussian_in.quats, 1 * sizeof(glm::fvec4));
    hipMemcpy(d_gaussian_in.quats, &quat, 1 * sizeof(glm::fvec4), hipMemcpyHostToDevice);
    hipMalloc(&d_gaussian_in.scales, 1 * sizeof(glm::fvec3));
    hipMemcpy(d_gaussian_in.scales, &scale, 1 * sizeof(glm::fvec3), hipMemcpyHostToDevice);

    // create primitive output
    cugsplat::preprocess::DevicePrimitiveOutImage2DGS d_gaussian_out;
    hipMalloc(&d_gaussian_out.opacities, 1 * sizeof(float));
    hipMalloc(&d_gaussian_out.means, 1 * sizeof(glm::fvec2));
    hipMalloc(&d_gaussian_out.conics, 1 * sizeof(glm::fvec3));
    hipMalloc(&d_gaussian_out.depths, 1 * sizeof(float));
    hipMalloc(&d_gaussian_out.radius, 1 * sizeof(glm::fvec2));
    d_gaussian_out.render_width = 800;
    d_gaussian_out.render_height = 600;
    d_gaussian_out.near_plane = 0.1f;
    d_gaussian_out.far_plane = 100.0f;
    d_gaussian_out.margin_factor = 100.0f;
    d_gaussian_out.filter_size = 0.1f;

    dim3 blockDim(1, 1, 1);
    dim3 gridDim(1, 1, 1);

    cugsplat::preprocess::PreprocessKernel<
        cugsplat::preprocess::DeviceSimplePinholeCameraEWA,
        cugsplat::preprocess::DevicePrimitiveInWorld3DGS,
        cugsplat::preprocess::DevicePrimitiveOutImage2DGS,
        false,
        1
    ><<<gridDim, blockDim>>>(
        d_camera,
        d_gaussian_in,
        d_gaussian_out,
        nullptr,
        nullptr
    );

    // free memory
    d_camera.free();
    d_gaussian_in.free();
    d_gaussian_out.free();
    return 0;
}