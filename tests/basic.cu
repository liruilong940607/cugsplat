#include <hip/hip_runtime.h>
#include <iostream>

#include "camera/model.h"
#include "camera/opencv_pinhole.h"
#include "primitive/gaussian.h"
#include "projection/image2dgs.h"
#include "projection/kernel.cuh"
#include "utils/types.h"

using namespace gsplat;

int main() {
    // create camera
    auto const focal_length = glm::fvec2(800.0f, 600.0f);
    auto const principal_point = glm::fvec2(400.0f, 300.0f);
    auto const world_to_camera_R = glm::fmat3(1.0f);
    auto const world_to_camera_t = glm::fvec3(0.0f);
    std::array<uint32_t, 2> resolution = {800, 600};

    auto const pose = SE3Mat{world_to_camera_t, world_to_camera_R};
    auto projector =
        BatchedOpencvPinholeProjection(1, &focal_length, &principal_point);
    auto camera = CameraModel(resolution, projector, pose);

    // create input gaussian
    auto const opacity = float(0.8f);
    auto const mean = glm::fvec3(0.0f, 0.0f, 1.0f);
    auto const quat = glm::fvec4(1.0f, 0.0f, 0.0f, 0.0f);
    auto const scale = glm::fvec3(1.0f, 1.0f, 1.0f);
    auto gaussian = BatchPrimitive3DGS(1, &opacity, &mean, &quat, &scale);

    // create operator
    OutputOperatorImage2DGS op;
    op.preprocess(camera, gaussian);
    std::cout << "Opacity: " << op.opacity << std::endl;
    std::cout << "Mean: " << op.mean.x << ", " << op.mean.y << std::endl;
    std::cout << "Conic: " << op.conic.x << ", " << op.conic.y << ", "
              << op.conic.z << std::endl;
    std::cout << "Depth: " << op.depth << std::endl;
    std::cout << "Radius: " << op.radius.x << ", " << op.radius.y << std::endl;

    // gsplat::device::DeviceSimplePinholeCameraEWA d_camera;
    // d_camera.n = 1;
    // hipMalloc(&d_camera.focal_lengths, 1 * sizeof(glm::fvec2));
    // hipMemcpy(d_camera.focal_lengths, &focal_length, 1 *
    // sizeof(glm::fvec2), hipMemcpyHostToDevice);
    // hipMalloc(&d_camera.principal_points, 1 * sizeof(glm::fvec2));
    // hipMemcpy(d_camera.principal_points, &principal_point, 1 *
    // sizeof(glm::fvec2), hipMemcpyHostToDevice);
    // hipMalloc(&d_camera.world_to_cameras_R, 1 * sizeof(glm::fmat3));
    // hipMemcpy(d_camera.world_to_cameras_R, &world_to_camera_R, 1 *
    // sizeof(glm::fmat3), hipMemcpyHostToDevice);
    // hipMalloc(&d_camera.world_to_cameras_t, 1 * sizeof(glm::fvec3));
    // hipMemcpy(d_camera.world_to_cameras_t, &world_to_camera_t, 1 *
    // sizeof(glm::fvec3), hipMemcpyHostToDevice);

    // // create primitive input
    // auto const opacity = float(0.8f);
    // auto const mean = glm::fvec3(0.0f, 0.0f, 1.0f);
    // auto const quat = glm::fvec4(1.0f, 0.0f, 0.0f, 0.0f);
    // auto const scale = glm::fvec3(1.0f, 1.0f, 1.0f);

    // gsplat::device::DevicePrimitiveInWorld3DGS d_gaussian_in;
    // d_gaussian_in.n = 1;
    // hipMalloc(&d_gaussian_in.opacity_ptr, 1 * sizeof(float));
    // hipMemcpy(d_gaussian_in.opacity_ptr, &opacity, 1 * sizeof(float),
    // hipMemcpyHostToDevice); hipMalloc(&d_gaussian_in.mean_ptr, 1 *
    // sizeof(glm::fvec3)); hipMemcpy(d_gaussian_in.mean_ptr, &mean, 1 *
    // sizeof(glm::fvec3), hipMemcpyHostToDevice);
    // hipMalloc(&d_gaussian_in.quat_ptr, 1 * sizeof(glm::fvec4));
    // hipMemcpy(d_gaussian_in.quat_ptr, &quat, 1 * sizeof(glm::fvec4),
    // hipMemcpyHostToDevice); hipMalloc(&d_gaussian_in.scale_ptr, 1 *
    // sizeof(glm::fvec3)); hipMemcpy(d_gaussian_in.scale_ptr, &scale, 1 *
    // sizeof(glm::fvec3), hipMemcpyHostToDevice);

    // // create primitive output
    // gsplat::device::DevicePrimitiveOutImage2DGS d_gaussian_out;
    // hipMalloc(&d_gaussian_out.opacities, 1 * sizeof(float));
    // hipMalloc(&d_gaussian_out.means, 1 * sizeof(glm::fvec2));
    // hipMalloc(&d_gaussian_out.conics, 1 * sizeof(glm::fvec3));
    // hipMalloc(&d_gaussian_out.depths, 1 * sizeof(float));
    // hipMalloc(&d_gaussian_out.radius, 1 * sizeof(glm::fvec2));
    // d_gaussian_out.render_width = 800;
    // d_gaussian_out.render_height = 600;
    // d_gaussian_out.near_plane = 0.1f;
    // d_gaussian_out.far_plane = 100.0f;
    // d_gaussian_out.margin_factor = 100.0f;
    // d_gaussian_out.filter_size = 0.1f;

    // dim3 blockDim(256, 1, 1);
    // dim3 gridDim(1, 1, 1);

    // gsplat::device::PreprocessKernel<
    //     gsplat::device::DeviceSimplePinholeCameraEWA,
    //     gsplat::device::DevicePrimitiveInWorld3DGS,
    //     gsplat::device::DevicePrimitiveOutImage2DGS,
    //     false,
    //     1
    // ><<<gridDim, blockDim>>>(
    //     d_camera,
    //     d_gaussian_in,
    //     d_gaussian_out,
    //     nullptr,
    //     nullptr
    // );
    // hipDeviceSynchronize();
    // hipError_t err = hipGetLastError();
    // if (err != hipSuccess) {
    //     std::cerr << "CUDA error: " << hipGetErrorString(err) <<
    //     std::endl;
    // }

    // // copy output data back to host
    // float opacity_out;
    // glm::fvec2 mean_out;
    // glm::fvec3 conic_out;
    // float depth_out;
    // glm::fvec2 radius_out;
    // hipMemcpy(&opacity_out, d_gaussian_out.opacities, sizeof(float),
    // hipMemcpyDeviceToHost); hipMemcpy(&mean_out, d_gaussian_out.means,
    // sizeof(glm::fvec2), hipMemcpyDeviceToHost); hipMemcpy(&conic_out,
    // d_gaussian_out.conics, sizeof(glm::fvec3), hipMemcpyDeviceToHost);
    // hipMemcpy(&depth_out, d_gaussian_out.depths, sizeof(float),
    // hipMemcpyDeviceToHost); hipMemcpy(&radius_out,
    // d_gaussian_out.radius, sizeof(glm::fvec2), hipMemcpyDeviceToHost);
    // std::cout << "Opacity: " << opacity_out << std::endl;
    // std::cout << "Mean: " << mean_out.x << ", " << mean_out.y <<
    // std::endl; std::cout << "Conic: " << conic_out.x << ", " <<
    // conic_out.y << ", " << conic_out.z << std::endl; std::cout << "Depth:
    // " << depth_out << std::endl; std::cout << "Radius: " << radius_out.x
    // << ", " << radius_out.y << std::endl;

    // // free memory
    // d_camera.free();
    // d_gaussian_out.free();

    // hipFree(d_gaussian_in.opacity_ptr);
    // hipFree(d_gaussian_in.mean_ptr);
    // hipFree(d_gaussian_in.quat_ptr);
    // hipFree(d_gaussian_in.scale_ptr);
    return 0;
}