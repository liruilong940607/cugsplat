#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "../helpers.h"
#include "tinyrend/common/math.h"
#include "tinyrend/common/vec.h"

using namespace tinyrend;

__global__ void test_rsqrt_kernel(float *x, float *y) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    y[idx] = rsqrt(x[idx]);
}

int test_rsqrt() {
    int fails = 0;

    {
        float h_x[2] = {4.0f, 1.0f};
        float h_y[2];
        float h_y_expected[2] = {0.5f, 1.0f};
        float *d_x, *d_y;
        hipMalloc((void **)&d_x, 2 * sizeof(float));
        hipMalloc((void **)&d_y, 2 * sizeof(float));
        hipMemcpy(d_x, h_x, 2 * sizeof(float), hipMemcpyHostToDevice);
        test_rsqrt_kernel<<<1, 2>>>(d_x, d_y);
        hipMemcpy(h_y, d_y, 2 * sizeof(float), hipMemcpyDeviceToHost);
        fails += CHECK((is_close<float, 2>(h_y, h_y_expected)), "");
        hipFree(d_x);
        hipFree(d_y);
    }

    return fails;
}

int main() {
    int fails = 0;

    fails += test_rsqrt();

    if (fails > 0) {
        printf("[common/math_cuda.cu] %d tests failed!\n", fails);
    } else {
        printf("[common/math_cuda.cu] All tests passed!\n");
    }

    return fails;
}